#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#include "implementation.cpp"

namespace gpu_array {
    template <class T>
    struct my_array {
        using data_t = T;

        T *data;
        int sizes[3];
        int strides[3];

        __device__ const T &operator()(int i, int j, int k) const {
            assert(i < sizes[0] && j < sizes[1] && k < sizes[2] && "out of bounds");
            return data[i * strides[0] + j * strides[1] + k * strides[2]];
        }

        __device__ T &operator()(int i, int j, int k) {
            assert(i < sizes[0] && j < sizes[1] && k < sizes[2] && "out of bounds");
            return data[i * strides[0] + j * strides[1] + k * strides[2]];
        }
    };

    template <typename T>
    my_array<T> gen_make_fortran_array_view(gen_fortran_array_descriptor *descriptor, my_array<T> *) {
        if (descriptor->rank != 3) {
            throw std::runtime_error("only 3-dimensional arrays are supported");
        }
        return my_array<T>{static_cast<T *>(descriptor->data),
            {descriptor->dims[0], descriptor->dims[1], descriptor->dims[2]},
            {1, descriptor->dims[0], descriptor->dims[0] * descriptor->dims[1]}};
    }

    template <typename T>
    gen_fortran_array_descriptor get_fortran_view_meta(my_array<T> *) {
        gen_fortran_array_descriptor descriptor;
        descriptor.type = cpp_bindgen::fortran_array_element_kind<T>::value;
        descriptor.rank = 3;
        descriptor.is_acc_present = true;
        return descriptor;
    }

    static_assert(cpp_bindgen::is_fortran_array_bindable<my_array<double>>::value, "");
    static_assert(cpp_bindgen::is_fortran_array_wrappable<my_array<double>>::value, "");
} // namespace gpu_array

namespace {
    __global__ void fill_array_kernel(gpu_array::my_array<double> a) {
        for (size_t i = 0; i < a.sizes[0]; ++i) {
            a(i, blockIdx.x, threadIdx.x) = i * 10000 + blockIdx.x * 100 + threadIdx.x;
        }
    }

    void fill_gpu_array_impl(gpu_array::my_array<double> a) { fill_array_kernel<<<a.sizes[1], a.sizes[0]>>>(a); }

    GEN_EXPORT_BINDING_WRAPPED_1(fill_gpu_array, fill_gpu_array_impl);
} // namespace
